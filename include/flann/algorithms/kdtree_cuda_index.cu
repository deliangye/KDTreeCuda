#include "hip/hip_runtime.h"
/***********************************************************************
* Software License Agreement (BSD License)
*
* Copyright 2017  Wolfgang Brandenburger (w.brandenburger@unibw.de). All rights reserved.
*
* THE BSD LICENSE
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*
* 1. Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS OR
* IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
* OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
* IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY DIRECT, INDIRECT,
* INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
* NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
* DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
* THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
* THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*************************************************************************/

#include "kdtree_cuda_index.h"
#include "kdtree_cuda_index.cuh"

#include <flann/algorithms/dist.h>

#include "tools/graphic.h"

namespace flann
{
	template <typename Distance>
	__global__
	void gpuSearch(gpuknnSearch<Distance> search_, int numberofqueries_)
	{
		for (int index = threadIdx.x + blockIdx.x*blockDim.x; index < numberofqueries_; index += gridDim.x*blockDim.x) {
			search_.getNeighbors(index);
		}
	}

	template void KDTreeCudaIndex<flann::L2<float>>::gpuMemCpyTrees();
	template void KDTreeCudaIndex<flann::L2<double>>::gpuMemCpyTrees();

	template <typename Distance> void KDTreeCudaIndex<Distance>::gpuMemCpyTrees()
	{
		HANDLE_ERROR(hipMalloc((void**)&devtreeroots, tree_roots_.size() * sizeof(int)));
		HANDLE_ERROR(hipMemcpy(devtreeroots, tree_roots_.data(), tree_roots_.size() * sizeof(int), hipMemcpyHostToDevice));

		HANDLE_ERROR(hipMalloc((void**)&devpool, std::pow(2, (std::ceil(std::log2(size_)) + 1)) * trees_ * sizeof(KDTreeCudaIndex<Distance>::Node)));
		HANDLE_ERROR(hipMemcpy(devpool, pool_.base, std::pow(2, (std::ceil(std::log2(size_)) + 1)) * trees_ * sizeof(KDTreeCudaIndex<Distance>::Node), hipMemcpyHostToDevice));
	}

	template void KDTreeCudaIndex<flann::L2<float>>::gpuMemCpyData();
	template void KDTreeCudaIndex<flann::L2<double>>::gpuMemCpyData();

	template <typename Distance> void KDTreeCudaIndex<Distance>::gpuMemCpyData()
	{
		HANDLE_ERROR(hipMalloc((void**)&devdataset, size_ * veclen_ * sizeof(ElementType)));
		HANDLE_ERROR(hipMemcpy(devdataset, points_[0], size_ * veclen_ * sizeof(ElementType), hipMemcpyHostToDevice));
	}

	template void KDTreeCudaIndex<flann::L2<float>>::knnSearchGpu(const Matrix<ElementType>& queries,
		Matrix<size_t>& indices,
		Matrix<DistanceType>& dists,
		size_t knn,
		const SearchParams& params) const;
	template void KDTreeCudaIndex<flann::L2<double>>::knnSearchGpu(const Matrix<ElementType>& queries,
		Matrix<size_t>& indices,
		Matrix<DistanceType>& dists,
		size_t knn,
		const SearchParams& params) const;

	template<typename Distance>
	void knnSearchGpuKernel(gpuknnSearch<Distance> search_, int numberofqueries_)
	{
		typedef typename Distance::ElementType ElementType;

		////hipDeviceProp_t prop;
		////int dev;		
		////HANDLE_ERROR(hipGetDevice(&dev));
		////printf("ID of current CUDA device: %d\n", dev);
		////HANDLE_ERROR(hipSetDevice(1)); // DETERMINE THE BEST GRAPHICCARD
		////HANDLE_ERROR(hipGetDevice(&dev));
		////printf("ID of current CUDA device: %d\n", dev);

		//HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, std::pow(2, 18)*search_.knn*search_.trees*std::log(search_.size) / std::log(2) * sizeof(ElementType)));
		//std::cout << std::pow(2, 17)*search_.knn*search_.trees*std::log(search_.size) / std::log(2) * sizeof(ElementType) << std::endl;

		//cudaPrintfInit();
		dim3 grid(28*32, 1);
		dim3 block(256, 1);
		gpuSearch<Distance> << <grid, block >> > (search_, numberofqueries_);
		//cudaPrintfDisplay(stdout, true);
		//cudaPrintfEnd();
	}

	template <typename Distance>
	void KDTreeCudaIndex<Distance>::knnSearchGpu(const Matrix<ElementType>& queries,
		Matrix<size_t>& indices,
		Matrix<DistanceType>& dists,
		size_t knn,
		const SearchParams& params) const
	{
		ElementType* devqueries;
		size_t* devindices;
		DistanceType* devdists;

		HANDLE_ERROR(hipMalloc((void**)&devqueries, queries.rows * queries.cols * sizeof(ElementType)));
		HANDLE_ERROR(hipMemcpy(devqueries, queries.ptr(), queries.rows * queries.cols * sizeof(ElementType), hipMemcpyHostToDevice));

		HANDLE_ERROR(hipMalloc((void**)&devindices, indices.rows * indices.cols * sizeof(size_t)));
		HANDLE_ERROR(hipMemcpy(devindices, indices.ptr(), indices.rows * indices.cols * sizeof(size_t), hipMemcpyHostToDevice));

		HANDLE_ERROR(hipMalloc((void**)&devdists, dists.rows * dists.cols * sizeof(DistanceType)));
		HANDLE_ERROR(hipMemcpy(devdists, dists.ptr(), dists.rows * dists.cols * sizeof(DistanceType), hipMemcpyHostToDevice));

		//size_t* devHeapNumber;
		//size_t* heapNumber(new size_t[queries.rows]);
		//for (int i = 0; i < queries.rows; i++) {
		//	heapNumber[i] = 0;
		//}
		//HANDLE_ERROR(hipMalloc((void**)&devHeapNumber, queries.rows * sizeof(size_t)));
		//HANDLE_ERROR(hipMemcpy(devHeapNumber, heapNumber, queries.rows * sizeof(size_t), hipMemcpyHostToDevice));

		if (std::is_same<Distance, flann::L2<ElementType>>::value) {
			typedef graphic::L2<ElementType> DistanceGpu;
			gpuknnSearch<DistanceGpu> search(devdataset, devpool, devtreeroots, devqueries, devindices, devdists/*, devHeapNumber*/, veclen_, size_, tree_roots_.size(), knn);
			knnSearchGpuKernel<DistanceGpu>(search, queries.rows);
			//versuchKernelCall<ElementType, DistanceType>(devtreeroots, trees_, veclen_, size_, devpool, devdataset);
		}
		else if (std::is_same<Distance, flann::L2_3D<ElementType>>::value) {
			typedef graphic::L2_3D<ElementType> DistanceGpu;
			gpuknnSearch<DistanceGpu> search(devdataset, devpool, devtreeroots, devqueries, devindices, devdists/*, devHeapNumber*/, veclen_, size_, tree_roots_.size(), knn);
			knnSearchGpuKernel<DistanceGpu>(search, queries.rows);
			//versuchKernelCall<ElementType,DistanceType>(devtreeroots, trees_, veclen_, size_, devpool, devdataset);
		}
		else if (std::is_same<Distance, flann::L2_Simple<ElementType>>::value) {
			typedef graphic::L2_Simple<ElementType> DistanceGpu;
			gpuknnSearch<DistanceGpu> search(devdataset, devpool, devtreeroots, devqueries, devindices, devdists/*, devHeapNumber*/, veclen_, size_, tree_roots_.size(), knn);
			knnSearchGpuKernel<DistanceGpu>(search, queries.rows);
			//versuchKernelCall<ElementType, DistanceType>(devtreeroots, trees_, veclen_, size_, devpool, devdataset);
		}

		HANDLE_ERROR(hipMemcpy(indices.ptr(), devindices, indices.rows * indices.cols * sizeof(size_t), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(dists.ptr(),devdists,dists.rows * dists.cols * sizeof(DistanceType), hipMemcpyDeviceToHost));

		//HANDLE_ERROR(hipMemcpy(heapNumber,devHeapNumber,queries.rows*sizeof(size_t),hipMemcpyDeviceToHost));

		HANDLE_ERROR(hipFree(devqueries));
		HANDLE_ERROR(hipFree(devindices));
		HANDLE_ERROR(hipFree(devdists));
	}

	template void KDTreeCudaIndex<flann::L2<float>>::gpuDestructor();
	template void KDTreeCudaIndex<flann::L2<double>>::gpuDestructor();

	template <typename Distance > void KDTreeCudaIndex<Distance>::gpuDestructor()
	{
		HANDLE_ERROR(hipFree(devpool));
		devpool = nullptr;
		HANDLE_ERROR(hipFree(devdataset));
		devdataset = nullptr;
		HANDLE_ERROR(hipFree(devtreeroots));
		devtreeroots = nullptr;
	}

	template void KDTreeCudaIndex<flann::L2<float>>::gpuFreeIndex();
	template void KDTreeCudaIndex<flann::L2<double>>::gpuFreeIndex();

	template <typename Distance > void KDTreeCudaIndex<Distance>::gpuFreeIndex()
	{
		HANDLE_ERROR(hipFree(devpool));
		devpool = nullptr;
		HANDLE_ERROR(hipFree(devtreeroots));
		devtreeroots = nullptr;
	}
}

